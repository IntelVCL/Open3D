#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/t/pipelines/kernel/ComputeTransformImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/t/pipelines/registration/RobustKernel.h"
#include "open3d/t/pipelines/registration/RobustKernelImpl.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;

template <typename scalar_t, typename func_t>
__global__ void ComputePosePointToPlaneKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum,
        func_t GetWeightFromRobustKernel) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_ij[6] = {0}, reduction[29] = {0};
    scalar_t r = 0;

    bool valid = GetJacobianPointToPlane<scalar_t>(
            workload_idx, source_points_ptr, target_points_ptr,
            target_normals_ptr, correspondence_indices, J_ij, r);

    scalar_t w = GetWeightFromRobustKernel(r);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        int i = 0;
        for (int j = 0; j < 6; ++j) {
            for (int k = 0; k <= j; ++k) {
                reduction[i] += J_ij[j] * w * J_ij[k];
                ++i;
            }
            reduction[21 + j] += J_ij[j] * w * r;
        }
        reduction[27] += r;
        reduction[28] += 1;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                      local_sum0, local_sum1,
                                                      local_sum2, global_sum);
}

void ComputePosePointToPlaneCUDA(const core::Tensor &source_points,
                                 const core::Tensor &target_points,
                                 const core::Tensor &target_normals,
                                 const core::Tensor &correspondence_indices,
                                 core::Tensor &pose,
                                 float &residual,
                                 int &inlier_count,
                                 const core::Dtype &dtype,
                                 const core::Device &device,
                                 const registration::RobustKernel &kernel) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePosePointToPlaneKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(), n,
                            global_sum_ptr, GetWeightFromRobustKernel);
                });
    });

    OPEN3D_CUDA_CHECK(hipDeviceSynchronize());

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

template <typename scalar_t, typename funct_t>
__global__ void ComputePoseColoredICPKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *source_colors_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const scalar_t *target_colors_ptr,
        const scalar_t *target_color_gradients_ptr,
        const int64_t *correspondence_indices,
        const scalar_t sqrt_lambda_geometric,
        const scalar_t sqrt_lambda_photometric,
        const int n,
        scalar_t *global_sum,
        funct_t op) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_G[6] = {0}, J_I[6] = {0}, reduction[29] = {0};
    scalar_t r_G = 0, r_I = 0;

    bool valid = GetJacobianColoredICP<scalar_t>(
            workload_idx, source_points_ptr, source_colors_ptr,
            target_points_ptr, target_normals_ptr, target_colors_ptr,
            target_color_gradients_ptr, correspondence_indices,
            sqrt_lambda_geometric, sqrt_lambda_photometric, J_G, J_I, r_G, r_I);

    scalar_t w_G = 1.0;  // op(r_G);
    scalar_t w_I = 1.0;  // op(r_I);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        reduction[0] = J_G[0] * w_G * J_G[0] + J_I[0] * w_I * J_I[0];
        reduction[1] = J_G[1] * w_G * J_G[0] + J_I[1] * w_I * J_I[0];
        reduction[2] = J_G[1] * w_G * J_G[1] + J_I[1] * w_I * J_I[1];
        reduction[3] = J_G[2] * w_G * J_G[0] + J_I[2] * w_I * J_I[0];
        reduction[4] = J_G[2] * w_G * J_G[1] + J_I[2] * w_I * J_I[1];
        reduction[5] = J_G[2] * w_G * J_G[2] + J_I[2] * w_I * J_I[2];
        reduction[6] = J_G[3] * w_G * J_G[0] + J_I[3] * w_I * J_I[0];
        reduction[7] = J_G[3] * w_G * J_G[1] + J_I[3] * w_I * J_I[1];
        reduction[8] = J_G[3] * w_G * J_G[2] + J_I[3] * w_I * J_I[2];
        reduction[9] = J_G[3] * w_G * J_G[3] + J_I[3] * w_I * J_I[3];
        reduction[10] = J_G[4] * w_G * J_G[0] + J_I[4] * w_I * J_I[0];
        reduction[11] = J_G[4] * w_G * J_G[1] + J_I[4] * w_I * J_I[1];
        reduction[12] = J_G[4] * w_G * J_G[2] + J_I[4] * w_I * J_I[2];
        reduction[13] = J_G[4] * w_G * J_G[3] + J_I[4] * w_I * J_I[3];
        reduction[14] = J_G[4] * w_G * J_G[4] + J_I[4] * w_I * J_I[4];
        reduction[15] = J_G[5] * w_G * J_G[0] + J_I[5] * w_I * J_I[0];
        reduction[16] = J_G[5] * w_G * J_G[1] + J_I[5] * w_I * J_I[1];
        reduction[17] = J_G[5] * w_G * J_G[2] + J_I[5] * w_I * J_I[2];
        reduction[18] = J_G[5] * w_G * J_G[3] + J_I[5] * w_I * J_I[3];
        reduction[19] = J_G[5] * w_G * J_G[4] + J_I[5] * w_I * J_I[4];
        reduction[20] = J_G[5] * w_G * J_G[5] + J_I[5] * w_I * J_I[5];

        reduction[21] = J_G[0] * w_G * r_G + J_I[0] * w_I * r_I;
        reduction[22] = J_G[1] * w_G * r_G + J_I[1] * w_I * r_I;
        reduction[23] = J_G[2] * w_G * r_G + J_I[2] * w_I * r_I;
        reduction[24] = J_G[3] * w_G * r_G + J_I[3] * w_I * r_I;
        reduction[25] = J_G[4] * w_G * r_G + J_I[4] * w_I * r_I;
        reduction[26] = J_G[5] * w_G * r_G + J_I[5] * w_I * r_I;

        reduction[27] = r_G * r_G + r_I * r_I;
        reduction[28] = 1;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                      local_sum0, local_sum1,
                                                      local_sum2, global_sum);
}

void ComputePoseColoredICPCUDA(const core::Tensor &source_points,
                               const core::Tensor &source_colors,
                               const core::Tensor &target_points,
                               const core::Tensor &target_normals,
                               const core::Tensor &target_colors,
                               const core::Tensor &target_color_gradients,
                               const core::Tensor &correspondence_indices,
                               core::Tensor &pose,
                               float &residual,
                               int &inlier_count,
                               const core::Dtype &dtype,
                               const core::Device &device,
                               const registration::RobustKernel &kernel,
                               const float &lambda_geometric) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t sqrt_lambda_geometric =
                static_cast<scalar_t>(sqrt(lambda_geometric));
        scalar_t sqrt_lambda_photometric =
                static_cast<scalar_t>(sqrt(1.0 - lambda_geometric));

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePoseColoredICPKernelCUDA<<<blocks, threads>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            source_colors.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            target_colors.GetDataPtr<scalar_t>(),
                            target_color_gradients.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(),
                            sqrt_lambda_geometric, sqrt_lambda_photometric, n,
                            global_sum.GetDataPtr<scalar_t>(), func_t);
                });
    });

    OPEN3D_CUDA_CHECK(hipDeviceSynchronize());

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
